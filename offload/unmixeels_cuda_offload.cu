// nvcc --compiler-options '-fPIC -lcudart -shared -lm -lR ' unmixeels_cuda_offload.cu -lcudart -o unmixeels_cuda_offload.so


#include <hip/hip_runtime.h>
#include <stdio.h>

#define MAX_THREADS_PER_BLOCK 256

__global__
void cuda_ddot(float *image_drifted_device, float *cor_device, 
		int nElems) 
{
	if (0 == threadIdx.x) {
		float mean = 0;int i;
		for (i=0; i<nElems; i++) 
			mean += image_drifted_device[blockIdx.x*nElems+i];
		mean /= nElems;
		for (i=0; i<nElems; i++) 
			image_drifted_device[blockIdx.x*nElems+i] -= mean;
		cor_device[blockIdx.x] = rnormf ( nElems, &image_drifted_device[blockIdx.x*nElems] );
	}
}


__global__ 
void cuda_cor(float *image_drifted_device, float *cor_device, 
		int nLines, int nElems, int fixed_line) 
{
	int i, j;
	float sum;

	if (threadIdx.x < nElems && blockIdx.x != fixed_line) {
		i = blockIdx.x*nElems + threadIdx.x;
		j = fixed_line*nElems + threadIdx.x;
		image_drifted_device[i] *=image_drifted_device[j];
	}
	__syncthreads();
	sum = 0;
	if (0 == threadIdx.x && blockIdx.x != fixed_line) {
		for (i=0;i<nElems;i++) {
			j = blockIdx.x*nElems + i;
			sum += image_drifted_device[j];
		}
		cor_device[blockIdx.x] *=  sum*cor_device[fixed_line];
	}
}

extern "C" 
void img_cor(double *image_drifted, double *cor, int *img_dims, int *fixed_line)
{
	float *image_drifted_device;
	float *cor_device;
	float *image_drifted_f, *cor_f;
	int i, blocksize; 
	const int nLines = img_dims[1];
	const int nElems = img_dims[0];
	const int nPixes = nLines * nElems;
	const int img_size = nPixes*sizeof(float);
	const int cor_size = nLines*sizeof(float);

	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, 0);

	if (nElems > prop.maxThreadsPerBlock) {
		printf("Error: correlation cannot be calculated for image with %d pixels per line. Max allowed number of pixels per line is %d.\n", nElems, prop.maxThreadsPerBlock);
	} else {
		image_drifted_f = (float *)malloc(img_size);
		for (i=0;i<nPixes;i++) {
			image_drifted_f[i] = image_drifted[i];
		}

		hipMalloc( (void**)&image_drifted_device, img_size ); 
		hipMalloc( (void**)&cor_device, cor_size ); 
		hipMemcpy( image_drifted_device, image_drifted_f, img_size, hipMemcpyHostToDevice ); 

		free(image_drifted_f);

		for (blocksize = 32; blocksize < nElems; blocksize*=2);
		dim3 dimBlock( blocksize, 1 );
		dim3 dimGrid( nLines, 1 );
		cuda_ddot<<<dimGrid, 1>>>(image_drifted_device, cor_device, 
						nElems);
		cuda_cor<<<dimGrid, dimBlock>>>(image_drifted_device, cor_device, 
						nLines, nElems, *fixed_line);
			
		cor_f = (float *)malloc(cor_size);
		hipMemcpy( cor_f, cor_device, cor_size, hipMemcpyDeviceToHost ); 
		hipFree( image_drifted_device );
		hipFree( cor_device );
		for (i=0;i<nLines;i++) {
			cor[i] = cor_f[i];
		}
		free(cor_f);
	}
}


__global__ 
void cuda_calc_kern(float *kern_func_device,
	int window, float mu, float sigma) 
{
	int i;
	__shared__ float scale;
	if (threadIdx.x < window ){
		kern_func_device[threadIdx.x] = 
			exp(-(threadIdx.x-mu)*(threadIdx.x-mu)/(2.*sigma*sigma));
	}
	__syncthreads();
	if (0 == threadIdx.x) {
		scale = 0;
		for (i=0;i<window;i++) {
			scale += kern_func_device[i];
		}
		scale = 1./scale;
	}
	__syncthreads();
	if (threadIdx.x < window ){
		kern_func_device[threadIdx.x] *= scale;
	}
}

__global__ 
void cuda_cp_ext(float *cnt_device, float *ext_device, 
	int nElems, int ext_nElems, int window) 
{
	int i,j,k,p;
	i = threadIdx.x + blockIdx.y * blockDim.x; 
	j = blockIdx.x*nElems;
	k = blockIdx.x*ext_nElems;
	p = (window - 1)/2;
	if (i<ext_nElems) {
		if (i < p) {
			ext_device[i+k] = cnt_device[j];
		} else if (i >= (nElems+p)){
			ext_device[i+k] = cnt_device[j+nElems-1];
		} else {
			ext_device[i+k] = cnt_device[i+j-p];
		} 
	}
}

__global__
void cuda_calc_smth(float *kern_func_device, float *ext_device, float *cnt_device, 
	int nElems, int ext_nElems, int window) 
{
	int i,j,k,p;
	float sum;
	i = threadIdx.x + blockIdx.y * blockDim.x; 
	j = blockIdx.x*nElems;
	k = blockIdx.x*ext_nElems;
	if (i < nElems) {
		sum = 0;
		for (p=0;p<window;p++) {
			sum += ext_device[i+p+k]*kern_func_device[p];
		}
		cnt_device[i+j] = sum;
	}
}


extern "C" 
void cuda_smth_cnt(double* cnt_out, double *cnt_in, int *cnt_dims, 
		int *window, double *alpha, int *tails) {
	float *cnt_f, *cnt_device, 
		*ext_device, *kern_func_device;
	int i, j, blocksize; 

	const int nLines = cnt_dims[1];
	const int nElems = cnt_dims[0];
	const int nPixes = nLines * nElems;
	const int cnt_size = nPixes*sizeof(float);
	const int kern_size = (*window)*sizeof(float);
	const int ext_nElems = *window+nElems;
	const int ext_nPixes = nLines * ext_nElems;
	const int ext_cnt_size = ext_nPixes*sizeof(float);

	float sigma, mu;

	if ((*window)<=0) {
		memcpy(cnt_out, cnt_in, nPixes*sizeof(double));
	} else {
		*window = (*window + 1)/2*2-1;
		sigma = .5 * (*window) / (*alpha);

		//Is there a better way when window is an even number?
		mu = floor((*window) / 2.);

		for (blocksize = 32; blocksize < *window; blocksize*=2);
		dim3 dimBlock( blocksize, 1 );
		dim3 dimGrid( nLines, 1 );
		hipMalloc( (void**)&kern_func_device, kern_size ); 
		cuda_calc_kern<<<1, dimBlock>>>(kern_func_device, *window, 
					mu, sigma);
		cnt_f = (float *)malloc(nPixes*sizeof(float));
		for (i=0;i<nPixes;i++) 
			cnt_f[i] = cnt_in[i];

		hipMalloc( (void**)&cnt_device, cnt_size ); 
		hipMalloc( (void**)&ext_device, ext_cnt_size ); 

		hipMemcpy( cnt_device, cnt_f, cnt_size, hipMemcpyHostToDevice );

		j = ext_nElems < MAX_THREADS_PER_BLOCK ? ext_nElems : MAX_THREADS_PER_BLOCK;
		for (blocksize = 32; blocksize < j; blocksize*=2);
		dimBlock.x = blocksize;
		dimGrid.y = ext_nElems / MAX_THREADS_PER_BLOCK + 1;
		cuda_cp_ext<<<dimGrid, dimBlock>>>(cnt_device, ext_device, 
			nElems, ext_nElems, *window);
		j = nElems < MAX_THREADS_PER_BLOCK ? nElems : MAX_THREADS_PER_BLOCK;
		for (blocksize = 32; blocksize < j; blocksize*=2);
		dimBlock.x = blocksize;
		dimGrid.y = nElems / MAX_THREADS_PER_BLOCK + 1;
		cuda_calc_smth<<<dimGrid, dimBlock>>>(kern_func_device, ext_device, cnt_device, 
			nElems, ext_nElems, *window);
		hipMemcpy( cnt_f, cnt_device, cnt_size, hipMemcpyDeviceToHost ); 
		hipFree( cnt_device );
		hipFree( ext_device );
		hipFree( kern_func_device );
		for (i=0;i<nPixes;i++) 
			cnt_out[i] = cnt_f[i];
		free(cnt_f);
	}
}


